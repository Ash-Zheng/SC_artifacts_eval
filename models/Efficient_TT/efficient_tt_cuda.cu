#include "hip/hip_runtime.h"
#include <assert.h>
#include <ATen/ATen.h>

#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
// #include <ATen/CUDAGeneratorImpl.h>
#include <ATen/TensorUtils.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "hashtbl_cuda_utils.cuh"
#include "tt_cuda_utils.cuh"
#include <iostream>

// #include "cub-1.8.0/cub/device/device_radix_sort.cuh"

#define WARP_SIZE 32
#define eps 1e-5
#define MAX_BATCH_SIZE 8192

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)


using namespace at;

float** group_map; // content point to result of intermediate result.
int32_t* group_flag; // indicate group_map is/not have update
int32_t* group_idx;
float *cache;
float *output_d;
int32_t *group_idx_h;


void init_cuda(
    int32_t device_id,
    const std::vector<int>& tt_q_shape,
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    int32_t batch_size,
    int32_t feature_dim
)
{
    hipSetDevice(device_id);

    if(!group_map)
    {
      int min_length = 370 * 370;
      int32_t cache_dim = tt_q_shape[0] * tt_q_shape[1] * tt_ranks[2];

      hipMalloc(&group_map, min_length*sizeof(float*));
      hipMalloc(&group_flag, min_length*sizeof(int32_t));
      // hipMalloc(&group_idx, sizeof(int32_t));  // GPU Mem
      hipMallocManaged(&group_idx, sizeof(int32_t));  // unified Mem
      hipMalloc(&cache, min_length * cache_dim * sizeof(float));
      hipMalloc(&output_d, batch_size * feature_dim * sizeof(float));
    }
}


inline void cuda_gemm_batched_fp32_fp32(
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float* alpha,
    void** a_array,
    int lda,
    void** b_array,
    int ldb,
    float* beta,
    void** c_array,
    int ldc,
    int batch_count) 
{
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipblasSetStream(handle, c10::cuda::getCurrentCUDAStream());
  hipblasGemmBatchedEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      a_array,
      HIP_R_32F,
      lda,
      b_array,
      HIP_R_32F,
      ldb,
      beta,
      c_array,
      HIP_R_32F,
      ldc,
      batch_count,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

inline void stream_cuda_gemm_batched_fp32_fp32(
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float* alpha,
    void** a_array,
    int lda,
    void** b_array,
    int ldb,
    float* beta,
    void** c_array,
    int ldc,
    int batch_count,
    hipStream_t stream) 
{
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipblasSetStream(handle, stream);
  hipblasGemmBatchedEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      a_array,
      HIP_R_32F,
      lda,
      b_array,
      HIP_R_32F,
      ldb,
      beta,
      c_array,
      HIP_R_32F,
      ldc,
      batch_count,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

__global__ void prepare_batch_gemm_pointers_3_core(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    int32_t index_length,
    int32_t output_length,
    int32_t cache_length,
    int32_t cache_dim,
    const int64_t* index,
    const int64_t* tt_p_shape,
    const int64_t* tt_q_shape,
    const int64_t* tt_ranks,
    float* tt_core_0,
    float* tt_core_1,
    float* tt_core_2,
    float* cache,
    float* result,
    float** __restrict__ a_ptr,
    float** __restrict__ b_ptr,
    float** __restrict__ c_ptr,
    float** group_map,
    int32_t* group_flag,
    int32_t* group_idx
    )
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(n >= batch_size)
    return;

  int idx_start = n;
  int idx_end = n + 1;

  int group;
  for(int i=idx_start;i<idx_end;i++){
    int idx = *(index + i);
    float tmp = float(idx)/tt_p_shape[2];
    group = floor(tmp);
    int I3 = idx % tt_p_shape[2];
    if(atomicCAS(group_flag + group, 0, 1)==0)
    {
      int cache_idx = atomicAdd(group_idx, 1);
      int I1 = floor(float(group)/tt_p_shape[1]);
      int I2 = group%tt_p_shape[1];
      // int I2 = floor(float(group)/tt_p_shape[0]);
      // int I1 = group%tt_p_shape[0];

      a_ptr[cache_idx] = tt_core_1 + I2 * tt_ranks[1] * tt_q_shape[1] * tt_ranks[2]; 
      b_ptr[cache_idx] = tt_core_0 + I1 * tt_q_shape[0] * tt_ranks[1];
      c_ptr[cache_idx] = cache + cache_idx * cache_dim;
      group_map[group] = cache + cache_idx * cache_dim;
    }
    a_ptr[cache_length + n] = tt_core_2 + I3 * (tt_q_shape[2] * tt_ranks[2]); 
    b_ptr[cache_length + n] = group_map[group]; // from cache
    c_ptr[cache_length + n] = result + n * output_length;
  }
}


__global__ void update_group_map(
  int32_t batch_size,
  int32_t cache_length,
  int32_t index_length,
  const int64_t* index,
  const int64_t* tt_p_shape,
  float** group_map,
  float** __restrict__ b_ptr
)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if(n >= batch_size)
    return;
  
  int idx = *(index + n);
  float tmp = float(idx)/tt_p_shape[2];
  int group = floor(tmp);
  // if(!b_ptr[cache_length + n])
  {
    b_ptr[cache_length + n] = group_map[group];
    // printf("%p\n",b_ptr[cache_length + n]);
  }
}

Tensor Efficient_TT_forward_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    const Tensor index,
    const std::vector<int>& tt_p_shape, //[i1,i2,i3]
    const std::vector<int>& tt_q_shape, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shape, //[i1,i2,i3]
    const Tensor tensor_q_shape, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    const std::vector<Tensor>& tt_cores
){
    auto output =
      at::zeros({batch_size, feature_dim}, tt_cores[0].options().dtype(at::kFloat)).to(at::kCUDA);
    
    int32_t index_length = index.sizes()[0];
    int32_t num_core = tt_p_shape.size();
    int32_t num_rank = tt_p_shape.size() + 1;
    int32_t cache_length = tt_p_shape[1] * tt_p_shape[2];
    int32_t cache_dim = tt_q_shape[0] * tt_q_shape[1] * tt_ranks[2];
    int32_t output_length = tt_q_shape[0] * tt_q_shape[1] * tt_q_shape[2];

    // // printf("\ncache_dim:%d,cache_length:%d,num_core:%d,num_rank:%d\n",cache_dim,cache_length,num_core,num_rank);

    auto a_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong)).to(at::kCUDA);
    auto b_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong)).to(at::kCUDA);
    auto c_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong)).to(at::kCUDA);
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();

    if(!group_map)
    {
      int min_length = 350 * 350;
      hipMalloc(&group_map, min_length*sizeof(float*));
      hipMalloc(&group_flag, min_length*sizeof(int32_t));
      // hipMalloc(&group_idx, sizeof(int32_t));  // GPU Mem
      hipMallocManaged(&group_idx, sizeof(int32_t));  // unified Mem
      hipMalloc(&cache, min_length * cache_dim * sizeof(float));
      hipMalloc(&output_d, batch_size * feature_dim * sizeof(float));
    }

    hipMemset(group_flag, 0, cache_length*sizeof(int32_t)); //set to zero
    hipMemset(group_idx, 0, sizeof(int32_t)); //set to zero

    // int32_t threads = 256;
    int32_t threads = (batch_size > 256 ? 256 : 32);
    int32_t num_blocks = (table_length + threads - 1) / threads; 
    prepare_batch_gemm_pointers_3_core<<<num_blocks, threads>>>(  // one thread lookup one row
      batch_size,
      table_length,
      feature_dim,
      index_length,
      output_length,
      cache_length,
      cache_dim,

      (const int64_t*)index.data_ptr(),
      (const int64_t*)tensor_p_shape.data_ptr(),
      (const int64_t*)tensor_q_shape.data_ptr(),
      (const int64_t*)tensor_ranks.data_ptr(),
      (float*)tt_cores[0].data_ptr(),
      (float*)tt_cores[1].data_ptr(),
      (float*)tt_cores[2].data_ptr(),
      cache,
      output_d,
      a_ptr,
      b_ptr,
      c_ptr,
      group_map,
      group_flag,
      group_idx
    );

    // int32_t *group_idx_h = (int32_t*)malloc(sizeof(int32_t));
    // hipMemcpy(group_idx_h, group_idx, sizeof(int32_t), hipMemcpyDeviceToHost);
    // printf("final group idx:%d\n",*group_idx_h);

    // use cuBlas batched gemm compute cache
    float alpha = 1.0;
    float beta = 0.0;
    // int batch_cnt = *group_idx_h;
    int batch_cnt = *group_idx;
    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      tt_q_shape[1]*tt_ranks[2], //n
      tt_q_shape[0],//m
      tt_ranks[1],//k
      &alpha,
      (void**)a_ptr,
      tt_q_shape[1]*tt_ranks[2], //n
      (void**)b_ptr,
      tt_ranks[1], // k
      &beta,
      (void**)c_ptr,
      tt_q_shape[1]*tt_ranks[2], // n
      batch_cnt
    );

    update_group_map<<<num_blocks, threads>>>(
      batch_size,
      cache_length,
      index_length,
      (const int64_t*)index.data_ptr(),
      (const int64_t*)tensor_p_shape.data_ptr(),
      group_map,
      b_ptr
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      tt_q_shape[2], //n
      tt_q_shape[0] * tt_q_shape[1],//m
      tt_ranks[2],//k
      &alpha,
      (void**)(a_ptr+cache_length),
      tt_q_shape[2], //n
      (void**)(b_ptr+cache_length),
      tt_ranks[2], // k
      &beta,
      (void**)(c_ptr+cache_length),
      tt_q_shape[2], // n
      batch_size
    );

    hipMemcpy((float*)output.data_ptr(), output_d, batch_size * feature_dim * sizeof(float), hipMemcpyDeviceToHost);

    return output;
}

//=============================================================================================================================
__global__ void prepare_batch_gemm_pointers_3_core_backward(
    int32_t N,
    const int64_t* __restrict__ index,
    const int64_t* tt_p_shape,

    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_cores_0,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_cores_1,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_cores_2,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores_0,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores_1,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores_2,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_0,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> d_output,
    int32_t* __restrict__ tt_idx,
    float** __restrict__ a_ptr,
    float** __restrict__ b_ptr,
    float** __restrict__ c_ptr,
    float** __restrict__ a0_ptr,
    float** __restrict__ b0_ptr,
    float** __restrict__ c0_ptr,
    float** __restrict__ a1_ptr,
    float** __restrict__ b1_ptr,
    float** __restrict__ c1_ptr) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    int32_t idx = *(index+n);

    float tmp = float(idx)/tt_p_shape[2];
    int group = floor(tmp);
    int I3 = idx % tt_p_shape[2];
    int I1 = floor(float(group)/tt_p_shape[1]);
    int I2 = group%tt_p_shape[1];

    tt_idx[0 * N + n] = I1;
    tt_idx[1 * N + n] = I2;
    tt_idx[2 * N + n] = I3;
    float* tr_0_ptr = (float*)&(tr_0[n][0]);
    float* d_output_ptr = (float*)&(d_output[n][0]);
    float* tt_cores_0_ptr = (float*)&(tt_cores_0[I1][0]);
    float* tt_cores_1_ptr = (float*)&(tt_cores_1[I2][0]);
    a_ptr[0 * N + n] = tt_cores_1_ptr;
    b_ptr[0 * N + n] = tt_cores_0_ptr;
    c_ptr[0 * N + n] = tr_0_ptr;

    a0_ptr[1 * N + n] = tr_0_ptr;
    b0_ptr[1 * N + n] = d_output_ptr;
    c0_ptr[1 * N + n] = (float*)&(tr_tt_cores_2[n][0]);
    a1_ptr[1 * N + n] = d_output_ptr;
    b1_ptr[1 * N + n] = (float*)&(tt_cores_2[I3][0]);
    c1_ptr[1 * N + n] = tr_0_ptr;

    a0_ptr[0 * N + n] = tt_cores_0_ptr;
    b0_ptr[0 * N + n] = tr_0_ptr;
    c0_ptr[0 * N + n] = (float*)&(tr_tt_cores_1[n][0]);
    a1_ptr[0 * N + n] = tr_0_ptr;
    b1_ptr[0 * N + n] = tt_cores_1_ptr;
    c1_ptr[0 * N + n] = (float*)&(tr_tt_cores_0[n][0]);
  }
}





__global__ void update_d_tt_cores_kernel(
    int32_t N,
    int32_t D,
    const int32_t* __restrict__ tt_idx,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> d_tt_cores) {
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n < N) {
    auto idx = __ldg(&tt_idx[n]);
    for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
      atomicAdd(&(d_tt_cores[idx][d]), 0.1 * tr_tt_cores[n][d]);
    }
  }
}

__global__ void update_tt_cores_sgd_kernel(
    int32_t B,
    int32_t D,
    float learning_rate,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> d_tt_cores,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_core
    ) {
  int32_t b = blockIdx.x * blockDim.y + threadIdx.y;
  if (b >= B) {
    return;
  }
  for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
    // tt_core[b][d] -= learning_rate * d_tt_cores[b][d];
    tt_core[b][d] -= d_tt_cores[b][d];
  }
}

__global__ void fused_update_tt_cores_sgd_kernel(
    int32_t B,
    int32_t D,
    float learning_rate,
    const int32_t* __restrict__ tt_idx,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_core
    ) {
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= B) {
    return;
  }
  learning_rate *= -1;
  auto idx = __ldg(&tt_idx[n]);
  for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
    auto delta = learning_rate * tr_tt_cores[n][d];
    atomicAdd(&(tt_core[idx][d]), delta);
  }
}

void Efficient_TT_backward_sgd_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    float learning_rate,

    const Tensor index,
    const std::vector<int>& tt_p_shapes, //[i1,i2,i3]
    const std::vector<int>& tt_q_shapes, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shapes, //[i1,i2,i3]
    const Tensor tensor_q_shapes, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    Tensor d_output,
    std::vector<Tensor>& tt_cores
)
{
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(d_output.get_device());
    int32_t T = 3;  //3
    int32_t batch_count = batch_size;

    std::vector<Tensor> d_tt_cores;
    std::vector<Tensor> tr_tt_cores;
    d_tt_cores.push_back(at::zeros_like(tt_cores[0]));
    d_tt_cores.push_back(at::zeros_like(tt_cores[1]));
    d_tt_cores.push_back(at::zeros_like(tt_cores[2]));
    
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[0].size(1)}, tt_cores[0].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[1].size(1)}, tt_cores[1].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[2].size(1)}, tt_cores[2].options()));

    std::vector<int32_t> m(T - 1);
    std::vector<int32_t> n(T - 1);
    std::vector<int32_t> k(T - 1);
    float alpha = 1.0;
    float beta = 0.0;
    int32_t m_ = tt_q_shapes[0]; 
    for (int32_t t = 0; t < T - 1; ++t) {
        m[t] = m_; //m[0]=j1 m[1]=j1*j2
        k[t] = tt_ranks[t + 1]; //k[0]=r1 k[1]=r2
        n[t] = tt_q_shapes[t + 1] * tt_ranks[t + 2]; //n[0]=j2*r2 n[1]=j3
        m_ = m_ * tt_q_shapes[t + 1];
    }

    std::vector<Tensor> tr;

    int64_t tr_size = tt_q_shapes[0] * tt_ranks[1];
    for (int32_t t = 0; t < T - 2; ++t) {
        tr_size = tr_size * tt_q_shapes[t + 1] * tt_ranks[t + 2] / tt_ranks[t + 1];
        tr.push_back(at::empty({batch_count, tr_size}, tt_cores[0].options()));
    }

    auto tt_idx =
      at::empty({T * batch_count}, tt_cores[0].options().dtype(at::kInt));
    auto a_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();
    auto a0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a0_ptr = (float**)a0_ptr_tensor.data_ptr<int64_t>();
    float** b0_ptr = (float**)b0_ptr_tensor.data_ptr<int64_t>();
    float** c0_ptr = (float**)c0_ptr_tensor.data_ptr<int64_t>();
    auto a1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a1_ptr = (float**)a1_ptr_tensor.data_ptr<int64_t>();
    float** b1_ptr = (float**)b1_ptr_tensor.data_ptr<int64_t>();
    float** c1_ptr = (float**)c1_ptr_tensor.data_ptr<int64_t>();

    int32_t start_idx = 0;
    int32_t end_idx = start_idx + batch_count;
    int32_t N = end_idx - start_idx;

    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;

    prepare_batch_gemm_pointers_3_core_backward<<<
        num_blocks,
        threads,
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
        N,
        (const int64_t*)index.data_ptr(),
        (const int64_t*)tensor_p_shapes.data_ptr(),

        tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_cores[1].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_cores[2].packed_accessor32<float, 2, RestrictPtrTraits>(),

        tr_tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[1].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[2].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        d_output.packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_idx.data_ptr<int32_t>(),
        a_ptr,
        b_ptr,
        c_ptr,
        a0_ptr,
        b0_ptr,
        c0_ptr,
        a1_ptr,
        b1_ptr,
        c1_ptr
    );
    
    cuda_gemm_batched_fp32_fp32(
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n[0],
        m[0],
        k[0],
        &alpha,
        (void**)&(a_ptr[0]),
        n[0],
        (void**)&(b_ptr[0]),
        k[0],
        &beta,
        (void**)&(c_ptr[0]),
        n[0],
        N);
    // for (int32_t t = 0; t < T - 2; ++t)
    // backward propagation

    for (int32_t t = T - 2; t >= 0; --t) {
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_N,
            HIPBLAS_OP_T,
            n[t],
            k[t],
            m[t],
            &alpha,
            (void**)&(b0_ptr[t * N]),
            n[t],
            (void**)&(a0_ptr[t * N]),
            k[t],
            &beta,
            (void**)&(c0_ptr[t * N]),
            n[t],
            N);
        int32_t D_0 = tt_cores[t + 1].size(1);
        int32_t tx_0 = std::min(1024, D_0);
        int32_t ty_0 = 1024 / tx_0;
        update_d_tt_cores_kernel<<<
            div_round_up(N, ty_0),
            dim3(tx_0, ty_0),
            0,
            c10::cuda::getCurrentCUDAStream()>>>(
            N,
            D_0,
            &(tt_idx.data_ptr<int32_t>()[(t + 1) * N]),
            tr_tt_cores[t + 1].packed_accessor32<float, 2, RestrictPtrTraits>(),
            d_tt_cores[t + 1].packed_accessor32<float, 2, RestrictPtrTraits>());
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            k[t],
            m[t],
            n[t],
            &alpha,
            (void**)&(b1_ptr[t * N]),
            n[t],
            (void**)&(a1_ptr[t * N]),
            n[t],
            &beta,
            (void**)&(c1_ptr[t * N]),
            k[t],
            N);
        if (t == 0) {
            int32_t D_1 = tt_cores[0].size(1);
            int32_t tx_1 = std::min(1024, D_1);
            int32_t ty_1 = 1024 / tx_1;
            update_d_tt_cores_kernel<<<
                div_round_up(N, ty_1),
                dim3(tx_1, ty_1),
                0,
                c10::cuda::getCurrentCUDAStream()>>>(
                N,
                D_1,
                &(tt_idx.data_ptr<int32_t>()[t * N]),
                tr_tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
                d_tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>());
        }
    } // for (int32_t t = T - 2; t >=0 ; --t)

    for (int32_t t = 0; t < T; ++t) {
        int32_t y_size = tt_cores[t].size(0);
        int32_t x_size = tt_cores[t].size(1);
        int32_t tx = std::min(1024, y_size);
        int32_t ty = 1024 / tx;
      
        update_tt_cores_sgd_kernel<<<
        div_round_up(x_size, ty),
        dim3(tx, ty),
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
          y_size,
          x_size,
          learning_rate, // hard code
          d_tt_cores[t].packed_accessor32<float, 2, RestrictPtrTraits>(),
          tt_cores[t].packed_accessor32<float, 2, RestrictPtrTraits>()
        );
    }
    
  return;
}


void Fused_Efficient_TT_backward_sgd_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    float learning_rate,

    const Tensor index,
    const std::vector<int>& tt_p_shapes, //[i1,i2,i3]
    const std::vector<int>& tt_q_shapes, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shapes, //[i1,i2,i3]
    const Tensor tensor_q_shapes, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    Tensor d_output,
    std::vector<Tensor>& tt_cores
)
{
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(d_output.get_device());
    int32_t T = 3;  //3
    int32_t batch_count = batch_size;

    std::vector<Tensor> tr_tt_cores;
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[0].size(1)}, tt_cores[0].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[1].size(1)}, tt_cores[1].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[2].size(1)}, tt_cores[2].options()));

    std::vector<int32_t> m(T - 1);
    std::vector<int32_t> n(T - 1);
    std::vector<int32_t> k(T - 1);
    float alpha = 1.0;
    float beta = 0.0;
    int32_t m_ = tt_q_shapes[0]; 
    for (int32_t t = 0; t < T - 1; ++t) {
        m[t] = m_; //m[0]=j1 m[1]=j1*j2
        k[t] = tt_ranks[t + 1]; //k[0]=r1 k[1]=r2
        n[t] = tt_q_shapes[t + 1] * tt_ranks[t + 2]; //n[0]=j2*r2 n[1]=j3
        m_ = m_ * tt_q_shapes[t + 1];
    }

    std::vector<Tensor> tr;

    int64_t tr_size = tt_q_shapes[0] * tt_ranks[1];
    for (int32_t t = 0; t < T - 2; ++t) {
        tr_size = tr_size * tt_q_shapes[t + 1] * tt_ranks[t + 2] / tt_ranks[t + 1];
        tr.push_back(at::empty({batch_count, tr_size}, tt_cores[0].options()));
    }

    auto tt_idx =
      at::empty({T * batch_count}, tt_cores[0].options().dtype(at::kInt));
    auto a_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();
    auto a0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a0_ptr = (float**)a0_ptr_tensor.data_ptr<int64_t>();
    float** b0_ptr = (float**)b0_ptr_tensor.data_ptr<int64_t>();
    float** c0_ptr = (float**)c0_ptr_tensor.data_ptr<int64_t>();
    auto a1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a1_ptr = (float**)a1_ptr_tensor.data_ptr<int64_t>();
    float** b1_ptr = (float**)b1_ptr_tensor.data_ptr<int64_t>();
    float** c1_ptr = (float**)c1_ptr_tensor.data_ptr<int64_t>();

    int32_t start_idx = 0;
    int32_t end_idx = start_idx + batch_count;
    int32_t N = end_idx - start_idx;

    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;

    prepare_batch_gemm_pointers_3_core_backward<<<
        num_blocks,
        threads,
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
        N,
        (const int64_t*)index.data_ptr(),
        (const int64_t*)tensor_p_shapes.data_ptr(),

        tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_cores[1].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_cores[2].packed_accessor32<float, 2, RestrictPtrTraits>(),

        tr_tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[1].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[2].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        d_output.packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_idx.data_ptr<int32_t>(),
        a_ptr,
        b_ptr,
        c_ptr,
        a0_ptr,
        b0_ptr,
        c0_ptr,
        a1_ptr,
        b1_ptr,
        c1_ptr
    );
    
    cuda_gemm_batched_fp32_fp32(
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n[0],
        m[0],
        k[0],
        &alpha,
        (void**)&(a_ptr[0]),
        n[0],
        (void**)&(b_ptr[0]),
        k[0],
        &beta,
        (void**)&(c_ptr[0]),
        n[0],
        N);

    for (int32_t t = T - 2; t >= 0; --t) {
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_N,
            HIPBLAS_OP_T,
            n[t],
            k[t],
            m[t],
            &alpha,
            (void**)&(b0_ptr[t * N]),
            n[t],
            (void**)&(a0_ptr[t * N]),
            k[t],
            &beta,
            (void**)&(c0_ptr[t * N]),
            n[t],
            N
            );
        
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            k[t],
            m[t],
            n[t],
            &alpha,
            (void**)&(b1_ptr[t * N]),
            n[t],
            (void**)&(a1_ptr[t * N]),
            n[t],
            &beta,
            (void**)&(c1_ptr[t * N]),
            k[t],
            N
            );
    } // for (int32_t t = T - 2; t >=0 ; --t)

    for (int32_t t = 0; t < T; ++t) {
        int32_t D_0 = tt_cores[t].size(1);
        int32_t tx_0 = std::min(1024, D_0);
        int32_t ty_0 = 1024 / tx_0;
        fused_update_tt_cores_sgd_kernel<<<
        div_round_up(N, ty_0),
        dim3(tx_0, ty_0),
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
          N,
          D_0,
          learning_rate, // hard code
          &(tt_idx.data_ptr<int32_t>()[t * N]),
          tr_tt_cores[t].packed_accessor32<float, 2, RestrictPtrTraits>(),
          tt_cores[t].packed_accessor32<float, 2, RestrictPtrTraits>()
        );
    }
    
  return;
}


// Extra_Eff_Fused ============================================================================

__global__ void Extra_Eff_prepare_batch_gemm_pointers_3_core_backward(
    int32_t unique_num,
    const int64_t* __restrict__ unique_index,
    const int64_t* tt_p_shape,

    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_cores_0,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_cores_1,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_cores_2,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores_0,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores_1,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores_2,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_0,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> d_output,
    int32_t* __restrict__ tt_idx,
    float** __restrict__ a_ptr,
    float** __restrict__ b_ptr,
    float** __restrict__ c_ptr,
    float** __restrict__ a0_ptr,
    float** __restrict__ b0_ptr,
    float** __restrict__ c0_ptr,
    float** __restrict__ a1_ptr,
    float** __restrict__ b1_ptr,
    float** __restrict__ c1_ptr) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < unique_num) {
    int32_t idx = *(unique_index+n);

    float tmp = float(idx)/tt_p_shape[2];
    int group = floor(tmp);
    int I3 = idx % tt_p_shape[2];
    int I1 = floor(float(group)/tt_p_shape[1]);
    int I2 = group%tt_p_shape[1];

    tt_idx[0 * unique_num + n] = I1;
    tt_idx[1 * unique_num + n] = I2;
    tt_idx[2 * unique_num + n] = I3;
    float* tr_0_ptr = (float*)&(tr_0[n][0]);
    float* d_output_ptr = (float*)&(d_output[n][0]);
    float* tt_cores_0_ptr = (float*)&(tt_cores_0[I1][0]);
    float* tt_cores_1_ptr = (float*)&(tt_cores_1[I2][0]);
    a_ptr[0 * unique_num + n] = tt_cores_1_ptr; //T1*T2
    b_ptr[0 * unique_num + n] = tt_cores_0_ptr;
    c_ptr[0 * unique_num + n] = tr_0_ptr;

    a0_ptr[1 * unique_num + n] = tr_0_ptr;
    b0_ptr[1 * unique_num + n] = d_output_ptr;
    c0_ptr[1 * unique_num + n] = (float*)&(tr_tt_cores_2[n][0]);
    a1_ptr[1 * unique_num + n] = d_output_ptr;
    b1_ptr[1 * unique_num + n] = (float*)&(tt_cores_2[I3][0]);
    c1_ptr[1 * unique_num + n] = tr_0_ptr;

    a0_ptr[0 * unique_num + n] = tt_cores_0_ptr; 
    b0_ptr[0 * unique_num + n] = tr_0_ptr;
    c0_ptr[0 * unique_num + n] = (float*)&(tr_tt_cores_1[n][0]);
    a1_ptr[0 * unique_num + n] = tr_0_ptr;
    b1_ptr[0 * unique_num + n] = tt_cores_1_ptr;
    c1_ptr[0 * unique_num + n] = (float*)&(tr_tt_cores_0[n][0]);
  }
}


__global__ void compute_unique_gradient(
  int batch_size,
  int feature_dim,
  const int64_t* inverse, 
  PackedTensorAccessor32<float, 2, RestrictPtrTraits> d_input,
  PackedTensorAccessor32<float, 2, RestrictPtrTraits> d_output
)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= batch_size)
    return;

  int idx = inverse[n];
  for(int i=0;i<feature_dim;i++)
  {
    atomicAdd(&(d_output[idx][i]), d_input[n][i]);
  }
}


__global__ void extra_fused_update_tt_cores_sgd_kernel(
    int32_t B,
    int32_t D,
    float learning_rate,
    const int32_t* __restrict__ tt_idx,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tr_tt_cores,
    PackedTensorAccessor32<float, 2, RestrictPtrTraits> tt_core
    ) {
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= B) {
    return;
  }
  auto idx = __ldg(&tt_idx[n]);
  for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
    auto delta = -1 * learning_rate * tr_tt_cores[n][d];
    atomicAdd(&(tt_core[idx][d]), delta);
    // tt_core[idx][d] -= learning_rate * tr_tt_cores[n][d];
  }
}


void Fused_Extra_Efficient_TT_backward_sgd_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    float learning_rate,

    const Tensor index,
    const std::vector<int>& tt_p_shapes, //[i1,i2,i3]
    const std::vector<int>& tt_q_shapes, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shapes, //[i1,i2,i3]
    const Tensor tensor_q_shapes, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    Tensor d_output,
    std::vector<Tensor>& tt_cores,
    Tensor sorted_idx,
    Tensor sorted_key
)
{
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(d_output.get_device());
    int32_t T = 3;  //3
    int32_t batch_count = batch_size;
    int32_t N = batch_size;
    int32_t unique_num = sorted_idx.size(0);


    auto unique_d = at::zeros({sorted_idx.size(0), d_output.size(1)}, tt_cores[0].options());

    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;

    compute_unique_gradient<<<num_blocks, threads>>>(
      batch_size,
      feature_dim,
      (const int64_t*)sorted_key.data_ptr(),
      d_output.packed_accessor32<float, 2, RestrictPtrTraits>(),
      unique_d.packed_accessor32<float, 2, RestrictPtrTraits>()
    );

    //===================================================================================================================
    std::vector<Tensor> tr_tt_cores;
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[0].size(1)}, tt_cores[0].options()));
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[1].size(1)}, tt_cores[1].options()));
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[2].size(1)}, tt_cores[2].options()));

    std::vector<int32_t> m(T - 1);
    std::vector<int32_t> n(T - 1);
    std::vector<int32_t> k(T - 1);
    float alpha = 1.0;
    float beta = 0.0;
    int32_t m_ = tt_q_shapes[0]; 
    for (int32_t t = 0; t < T - 1; ++t) {
        m[t] = m_; //m[0]=j1 m[1]=j1*j2
        k[t] = tt_ranks[t + 1]; //k[0]=r1 k[1]=r2
        n[t] = tt_q_shapes[t + 1] * tt_ranks[t + 2]; //n[0]=j2*r2 n[1]=j3
        m_ = m_ * tt_q_shapes[t + 1];
    }

    std::vector<Tensor> tr;

    int64_t tr_size = tt_q_shapes[0] * tt_ranks[1];
    for (int32_t t = 0; t < T - 2; ++t) {
        tr_size = tr_size * tt_q_shapes[t + 1] * tt_ranks[t + 2] / tt_ranks[t + 1];
        tr.push_back(at::empty({unique_num, tr_size}, tt_cores[0].options()));
    }

    auto tt_idx =
      at::empty({T * unique_num}, tt_cores[0].options().dtype(at::kInt));
    auto a_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();
    auto a0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a0_ptr = (float**)a0_ptr_tensor.data_ptr<int64_t>();
    float** b0_ptr = (float**)b0_ptr_tensor.data_ptr<int64_t>();
    float** c0_ptr = (float**)c0_ptr_tensor.data_ptr<int64_t>();
    auto a1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a1_ptr = (float**)a1_ptr_tensor.data_ptr<int64_t>();
    float** b1_ptr = (float**)b1_ptr_tensor.data_ptr<int64_t>();
    float** c1_ptr = (float**)c1_ptr_tensor.data_ptr<int64_t>();

    threads = (unique_num > 256 ? 256 : 32);
    num_blocks = (unique_num + threads - 1) / threads;

    Extra_Eff_prepare_batch_gemm_pointers_3_core_backward<<<
        num_blocks,
        threads,
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
        unique_num,
        (const int64_t*)sorted_idx.data_ptr(),
        (const int64_t*)tensor_p_shapes.data_ptr(),

        tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_cores[1].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_cores[2].packed_accessor32<float, 2, RestrictPtrTraits>(),

        tr_tt_cores[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[1].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[2].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tr[0].packed_accessor32<float, 2, RestrictPtrTraits>(),
        unique_d.packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_idx.data_ptr<int32_t>(),
        a_ptr,
        b_ptr,
        c_ptr,
        a0_ptr,
        b0_ptr,
        c0_ptr,
        a1_ptr,
        b1_ptr,
        c1_ptr
    );
    
    cuda_gemm_batched_fp32_fp32(
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n[0],
        m[0],
        k[0],
        &alpha,
        (void**)&(a_ptr[0]),
        n[0],
        (void**)&(b_ptr[0]),
        k[0],
        &beta,
        (void**)&(c_ptr[0]),
        n[0],
        unique_num);

// //======================================================
    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      n[1],
      k[1],
      m[1],
      &alpha,
      (void**)&(b0_ptr[unique_num]),
      n[1],
      (void**)&(a0_ptr[unique_num]),
      k[1],
      &beta,
      (void**)&(c0_ptr[unique_num]),
      n[1],
      unique_num
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      k[1],
      m[1],
      n[1],
      &alpha,
      (void**)&(b1_ptr[unique_num]),
      n[1],
      (void**)&(a1_ptr[unique_num]),
      n[1],
      &beta,
      (void**)&(c1_ptr[unique_num]),
      k[1],
      unique_num
    );
//=========================================================

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      n[0],
      k[0],
      m[0],
      &alpha,
      (void**)&(b0_ptr[0]),
      n[0],
      (void**)&(a0_ptr[0]),
      k[0],
      &beta,
      (void**)&(c0_ptr[0]),
      n[0],
      unique_num
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      k[0],
      m[0],
      n[0],
      &alpha,
      (void**)&(b1_ptr[0]),
      n[0],
      (void**)&(a1_ptr[0]),
      n[0],
      &beta,
      (void**)&(c1_ptr[0]),
      k[0],
      unique_num
    );

//=========================================================
  // return (a + b - 1) / b;
  for (int32_t t = 0; t < T; ++t) {
      int32_t D_0 = tt_cores[t].size(1);
      int32_t tx_0 = std::min(1024, D_0);
      int32_t ty_0 = 1024 / tx_0;
      extra_fused_update_tt_cores_sgd_kernel<<<
      div_round_up(unique_num, ty_0),
      dim3(tx_0, ty_0),
      0,
      c10::cuda::getCurrentCUDAStream()>>>(
        unique_num,
        D_0,
        learning_rate,
        &(tt_idx.data_ptr<int32_t>()[t * unique_num]),
        tr_tt_cores[t].packed_accessor32<float, 2, RestrictPtrTraits>(),
        tt_cores[t].packed_accessor32<float, 2, RestrictPtrTraits>()
      );
  }
  return;
}




